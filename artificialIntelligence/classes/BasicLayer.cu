#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <sstream>
#include <fstream>

#include <coreutils/classes/matrixes/Matrix3D.cpp>
#include <artificialIntelligence/functions/activationFunctions.hpp>
#include <artificialIntelligence/classes/BasicLayer.hpp>
#include <artificialIntelligence/classes/BasicWeight.hpp>

using namespace std;
using namespace coreutils::classes::matrixes;
using namespace artificialIntelligence::classes;
using namespace artificialIntelligence::functions::activation;


// a weight list of lists of lists of Matrix3D


BasicLayer::BasicLayer (Matrix3D* layerMatrix, Matrix3D* biasMatrix, BasicWeight* weights) {
   this->layerMatrix = new Matrix3D(layerMatrix->getLength(), layerMatrix->getWidth(), layerMatrix->getHeight());
   this->layerMatrix->setMatrix(layerMatrix);
   if (biasMatrix != nullptr) {
      this->biasMatrix = new Matrix3D(biasMatrix->getLength(), biasMatrix->getWidth(), biasMatrix->getHeight());
      this->biasMatrix->setMatrix(biasMatrix);
   } else {
      this->biasMatrix = nullptr;
   }
   this->weights = weights;
   this->next = nullptr;
   this->prev = nullptr;
}


BasicLayer::BasicLayer (int length, int width, int height) {
   this->layerMatrix = new Matrix3D (length, width, height);
   this->layerMatrix->randomize();
   this->biasMatrix = nullptr;
   this->weights = nullptr;
   this->next = nullptr;
   this->prev = nullptr;
}


BasicLayer::BasicLayer () {
   this->layerMatrix = nullptr;
   this->biasMatrix = nullptr;
   this->weights = nullptr;
   this->next = nullptr;
   this->prev = nullptr;
}


BasicLayer::~BasicLayer () { 
   if (this->layerMatrix != nullptr) {
      delete this->layerMatrix;
   }
   if (this->biasMatrix != nullptr) {
      delete this->biasMatrix;
   }
   if (this->weights != nullptr) {
      delete this->weights;
   }
   if (this->prev != nullptr) {
      this->prev->next = this->next;
   }
}

// 
// BasicLayer::~BasicLayer () {
//    if (this->next != nullptr) {
//       delete this->next;
//    }
// }


int BasicLayer::print (bool printBias, bool printWeights, int depth) {
   if (this->layerMatrix != nullptr) {
      std::cout << "\n\nCurrent Index: " << depth << '\n';
      std::cout << "Layer Matrix: \n";
      this->layerMatrix->printMatrix();
   } else {
      std::cout << "No layer found!\n";
      return depth;
   }
   if (printBias) {
      if (this->biasMatrix != nullptr) {
         std::cout << "Bias Matrix: \n";
         this->biasMatrix->printMatrix();
      } else {
         std::cout << "No biases found!\n";
      }
   }
   if (printWeights) {
      if (this->weights != nullptr) {
         this->weights->print();
      } else {
         std::cout << "No weights found!\n";
      }
   }
   if (this->next == nullptr) {
      return depth;
   }
   return this->next->print(printBias, printWeights, depth + 1);;
}  

//if it hits the end, it adds a new one to the back of the list and then
//returns the newly added node, along with a way to tell the previous node the weights needed

BasicLayer* BasicLayer::add (Matrix3D* layerMatrix, Matrix3D* biasMatrix, BasicWeight* weights) {
	
   if (next == nullptr) {
      this->next = new BasicLayer (layerMatrix, nullptr, nullptr);
      this->next->setPrev(this);
      if (this->biasMatrix == nullptr) {
         // std::cout << this->next->layerMatrix->getLength() << " " << this->next->layerMatrix->getWidth() << " " << this->next->layerMatrix->getHeight();
         this->biasMatrix = new Matrix3D(this->next->layerMatrix->getLength(), this->next->layerMatrix->getWidth(), this->next->layerMatrix->getHeight());
         this->biasMatrix->randomize(-0.05, 0.05);
      } else {
         this->biasMatrix->setMatrix(biasMatrix);
      }

		this->weights = this->newWeight(this, this->next);
      return this;
   }
   this->next->add(layerMatrix, biasMatrix, weights);
   return this;
}

BasicWeight* BasicLayer::newWeight(BasicLayer* firstLayer, BasicLayer* secondLayer) {
	return new BasicWeight (
		firstLayer->getLayerMatrix()->getLength(), 
		firstLayer->getLayerMatrix()->getWidth(),
		firstLayer->getLayerMatrix()->getHeight(),
		secondLayer->getLayerMatrix()->getLength(),
		secondLayer->getLayerMatrix()->getWidth(),
		secondLayer->getLayerMatrix()->getHeight());
}


BasicLayer* BasicLayer::add (BasicLayer* layer) {
   if (this->next == nullptr) {
      this->next = layer;
   } else {
      this->next = this->next->add(layer);
   }
   return this;
}


void artificialIntelligence::classes::BasicLayer::calculateAndUpdateAll () {
   if (this->next == nullptr) {
      return;
   }
   this->calculateAndUpdateSingle();
   this->next->calculateAndUpdateAll();
}


void BasicLayer::calculateAndUpdateSingle () {
   // start with the first node, and add all of the values to a node then sigmoid

   Matrix3D* nextLayer = this->next->getLayerMatrix();
   Matrix3D* outputs = new Matrix3D (nextLayer->getLength(), nextLayer->getWidth(), nextLayer->getHeight());
   if (isnan(*outputs->getData(0, 0, 0))) {
      std::cout << "null init";
      exit (0);
   }
   // start at start layer, then go to the end layer
   
   // declaring temp variables
   Matrix3D* weights;
   float activation = 0;

   // loop through every weight matrix
   // std::cout << "[" << this->layerMatrix->getLength() << "] " << "[" << this->layerMatrix->getWidth() << "] " << "[" << this->layerMatrix->getHeight() << "]   " 
   // << "[" << nextLayer->getLength() << "] " << "[" << nextLayer->getWidth() << "] " << "[" << nextLayer->getHeight() << "]" << "\n\n";
   for (int fl = 0; fl < this->layerMatrix->getLength(); fl++) {
      for (int fw = 0; fw < this->layerMatrix->getWidth(); fw++) {
         for (int fh = 0; fh < this->layerMatrix->getHeight(); fh++) {
            
            // making the activation start at the bias point
            // this returns the matrix for each node
            // now the matrix needs to be factored into each 
            weights = this->weights->getWeightMatrix(fl, fw, fh);
         
            // std::cout << "[" << fl << "] " << "[" << fw << "] " << "[" << fh << "] " << '\n';

            // if (fw == 1) {
            //    if (weights == nullptr) {
            //       this->weights->print();
            //       layerMatrix->printMatrix();
            //       std::cout << "error";
            //       exit (0);
            //    }
            //    weights->printMatrix();
            // }

            for (int sl = 0; sl < nextLayer->getLength(); sl++) {
               for (int sw = 0; sw < nextLayer->getWidth(); sw++) {
                  for (int sh = 0; sh < nextLayer->getHeight(); sh++) {
                     
                     // if (isnan(activation)) {
                     //    std::cout << this->layerMatrix->getData(fl, fw, fh) << " ";
                     //    std::cout << weights->getData(sl, sw, sh) << " ";
                     //    std::cout << outputs->getData(sl, sw, sh) << " ";
                     //    outputs->printMatrix();
                     //    std::cout << "\n" << sl << " " << sw << " " << sh;
                     //    std::cout << "\nactivation\n";
                     //    exit (0);
                     // }
                     
                     activation = *this->layerMatrix->getData(fl, fw, fh) * *weights->getData(sl, sw, sh) + *outputs->getData(sl, sw, sh);

                     // std::cout << "[" << fl << "] " << "[" << fw << "] " << "[" << fh << "]   " << "[" << sl << "] " << "[" << sw << "] " << "[" << sh << "]" << '\n';
                        
                     // std::cout << this->layerMatrix->getData(fl, fw, fh) << "    " << weights->getData(sl, sw, sh) << "   " << activation <<  '\n';
                     outputs->insert(activation, sl, sw, sh);
                  }
               }
            }
         }
      }
   } 

   // adds the bias and takes the sigmoid
   for (int sl = 0; sl < nextLayer->getLength(); sl++) {
      for (int sw = 0; sw < nextLayer->getWidth(); sw++) {
         for (int sh = 0; sh < nextLayer->getHeight(); sh++) {
            activation = sigmoid(*outputs->getData(sl, sw, sh) + *this->biasMatrix->getData(sl, sw, sh));
            // std::cout << outputs->getData(sl, sw, sh) << '\n';
            outputs->insert(activation, sl, sw, sh);
         }
      }
   }

   // set the next matrix to the layer that was just found
   this->next->setLayerMatrix (outputs);
   delete outputs;

}


void BasicLayer::setPrev (BasicLayer* prev) {
   if (this->prev != nullptr) {
      delete this->prev;
   }
   this->prev = prev;
}


Matrix3D* BasicLayer::getLayerMatrix () {
   return this->layerMatrix;
}


void BasicLayer::setLayerMatrix (Matrix3D* layerMatrix) {
   if (this->layerMatrix == nullptr) {
      this->layerMatrix = new Matrix3D (layerMatrix->getLength(), layerMatrix->getWidth(), layerMatrix->getHeight());
   }
   this->layerMatrix->setMatrix(layerMatrix);
}


Matrix3D* BasicLayer::getWeights (int length, int width, int height) {
   return this->weights->getWeightMatrix(length, width, height);
}

// 
// void BasicLayer::setWeights (BasicWeight* weights) {
//    if (this->weights != nullptr) {
//       this->weights->setMatrix(&weights);
//    } else {
//       this->weights->setMatrix (weights);
//       this->weights = weights;
//    }
// }


Matrix3D* BasicLayer::getBias () {
   return this->biasMatrix;
}


void BasicLayer::setBiasMatrix (Matrix3D* biasMatrix) {
   if (this->biasMatrix == nullptr) {
      this->biasMatrix = new Matrix3D (biasMatrix->getLength(), biasMatrix->getWidth(), biasMatrix->getHeight());
   }
   this->biasMatrix->setMatrix(biasMatrix);
}


BasicLayer* BasicLayer::getLast () {
   if (this->next == nullptr) {
      return this;
   }
   return this->next->getLast();
}


BasicLayer* BasicLayer::getNext () {
   return this->next;
}


BasicLayer* BasicLayer::getPrev () {
   return this->prev;
}


void BasicLayer::toFile (std::ofstream* outputFile) {
   *outputFile << this->layerMatrix->getLength() << ',' << this->layerMatrix->getWidth() << ',' << this->layerMatrix->getHeight() << '\n';

   // print bias values
   if (this->biasMatrix == nullptr) {
      return;
   }
   *outputFile << this->biasMatrix->getLength() << ',' << this->biasMatrix->getWidth() << ',' << this->biasMatrix->getHeight() << '\n';
   for (int i = 0; i < this->biasMatrix->getLength(); i++) {
      for (int j = 0; j < this->biasMatrix->getWidth(); j++) {
         for (int k = 0; k < this->biasMatrix->getHeight(); k++) {
            *outputFile << *this->biasMatrix->getData(i, j, k) << ',';
         }
      }
   }

   outputFile->seekp((int) outputFile->tellp() - 1);
   outputFile->write("\n", 1);

   if (this->weights == nullptr) {
      return;
   }

   // print weight values
   *outputFile << this->layerMatrix->getLength() << ',' << this->layerMatrix->getWidth() << ',' << this->layerMatrix->getHeight() << ',';
   *outputFile << this->biasMatrix->getLength() << ',' << this->biasMatrix->getWidth() << ',' << this->biasMatrix->getHeight() << '\n';
   for (int l = 0; l < this->layerMatrix->getLength(); l++) {
      for (int w = 0; w < this->layerMatrix->getWidth(); w++) {
         for (int h = 0; h < this->layerMatrix->getHeight(); h++) {
            for (int l2 = 0; l2 < this->biasMatrix->getLength(); l2++) {
               for (int w2 = 0; w2 < this->biasMatrix->getWidth(); w2++) {
                  for (int h2 = 0; h2 < this->biasMatrix->getHeight(); h2++) {
                     *outputFile << *this->weights->getData(l, w, h, l2, w2, h2) << ',';
                  }
               }
            }
         }
      }
   }

   outputFile->seekp((int) outputFile->tellp() - 1);
   outputFile->write("\n", 1); 

   if (this->next == nullptr) {
      return;
   }
   this->next->toFile(outputFile);
}


BasicLayer* BasicLayer::loadFromFile (std::ifstream* inputFile, BasicLayer* prev) {
   BasicLayer* layer = new BasicLayer ();
   std::string line;
   getline (*inputFile, line);
   std::stringstream lineStream;
   lineStream << line;
   std::string value;
   getline(lineStream, value, ',');
   int layerLength = stoi(value);
   getline(lineStream, value, ',');
   int layerWidth = stoi(value);
   getline(lineStream, value, ',');
   int layerHeight = stoi(value);
   Matrix3D* layerMatrix = new Matrix3D (layerLength, layerWidth, layerHeight);
   layer->layerMatrix = layerMatrix;
   layer->prev = prev;
      // std::cout << layerLength << " " << layerWidth << " " << layerHeight;

   lineStream.str(std::string());
   lineStream.clear();
   getline (*inputFile, line);
   lineStream << line;

   if (inputFile->eof()) {
      layer->biasMatrix = nullptr;
      layer->weights = nullptr;
      return layer;
   }

   getline(lineStream, value, ',');
   int biasLength = stoi(value);
   getline(lineStream, value, ',');
   int biasWidth = stoi(value);
   getline(lineStream, value, ',');
   int biasHeight = stoi(value);
   Matrix3D* biasMatrix = new Matrix3D (biasLength, biasWidth, biasHeight);
   layer->biasMatrix = biasMatrix;

   lineStream.str(std::string());
   lineStream.clear();
   getline (*inputFile, line);
   lineStream << line;
   for (int i = 0; i < layer->biasMatrix->getLength(); i++) {
      for (int j = 0; j < layer->biasMatrix->getWidth(); j++) {
         for (int k = 0; k < layer->biasMatrix->getHeight(); k++) {
            std::getline(lineStream, value, ',');
            layer->biasMatrix->insert (stod(value), i, j, k);
         }
      }
   }

   getline (*inputFile, line);

   if (inputFile->eof()) {
      layer->weights = nullptr;
      return layer;
   }

   BasicWeight* weights = new BasicWeight (
      layer->layerMatrix->getLength(), 
      layer->layerMatrix->getWidth(), 
      layer->layerMatrix->getHeight(), 
      layer->biasMatrix->getLength(), 
      layer->biasMatrix->getWidth(), 
      layer->biasMatrix->getHeight()
   );

   
   lineStream.str(std::string());
   lineStream.clear();
   getline (*inputFile, line);
   lineStream << line;
   // exit (0);
   for (int l = 0; l < layerLength; l++) {
      for (int w = 0; w < layerWidth; w++) {
         for (int h = 0; h < layerHeight; h++) {
            for (int l2 = 0; l2 < biasLength; l2++) {
               for (int w2 = 0; w2 < biasWidth; w2++) {
                  for (int h2 = 0; h2 < biasHeight; h2++) {
                     std::getline(lineStream, value, ',');
                     weights->insert(stod(value), l, w, h, l2, w2, h2);
                  }
               }
            }
         }
      }
   }

   layer->weights = weights;
   layer->next = BasicLayer::loadFromFile (inputFile, layer);

   return layer;
}
