#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <sstream>
#include <fstream>
#include <cmath>
#include <unistd.h>

#include <coreutils/classes/matrixes/Matrix3D.cuh>
#include <coreutils/functions/debug/print.hpp>
#include <coreutils/util/time.hpp>
#include <coreutils/util/cudaErrors.cuh>

#include "../../functions/activationFunctions.cuh"
#include "../layers/BasicLayer.cuh"
#include "../weights/BasicWeight.cuh"

using namespace std;
using namespace coreutils::classes::matrixes;
using namespace coreutils::functions::debug;
using namespace artificialIntelligence::classes;
using namespace artificialIntelligence::functions::activation;

#define MAX_BLOCK_SIZE 8192

BasicLayer::BasicLayer (Matrix3D* layerMatrix, Matrix3D* biasMatrix, BasicWeight* weights) {
   this->layerMatrix = new Matrix3D(layerMatrix->getLength(), layerMatrix->getWidth(), layerMatrix->getHeight());
   this->getLayer()->setMatrix(layerMatrix);
	this->biasMatrixes = new Matrix3D*[1];
	this->biasMatrixes[0] = nullptr;
   if (biasMatrix != nullptr) {
		std::cout << "here\n";
      this->biasMatrixes[0] = new Matrix3D(biasMatrix->getLength(), biasMatrix->getWidth(), biasMatrix->getHeight());
      this->getBias()->setMatrix(biasMatrix);
   }
	this->weights = (WeightBase**) new BasicWeight*[1];
   this->weights[0] = weights;
   this->next = (LayerBase**) new BasicLayer*[1];
	this->next[0] = nullptr;
   this->prev = (LayerBase**) new BasicLayer*[1];
	this->prev[0] = nullptr;
}


BasicLayer::BasicLayer (int length, int width, int height) {
   this->layerMatrix = new Matrix3D (length, width, height);
   this->getLayer()->randomize();
   this->biasMatrixes = new Matrix3D*[1];
	this->biasMatrixes[0] = nullptr;
   this->weights = (WeightBase**) new BasicWeight*[1];
   this->next = (LayerBase**) new BasicLayer*[1];
	this->next[0] = nullptr;
   this->prev = (LayerBase**) new BasicLayer*[1];
	this->prev[0] = nullptr;
}


BasicLayer::BasicLayer () {
   this->layerMatrix = nullptr;
   this->biasMatrixes = new Matrix3D*[1];
	this->biasMatrixes[0] = nullptr;
   this->weights = (WeightBase**) new BasicWeight*[1];
	this->weights[0] = nullptr;
   this->next = (LayerBase**) new BasicLayer*[1];
	this->next[0] = nullptr;
   this->prev = (LayerBase**) new BasicLayer*[1];
	this->prev[0] = nullptr;
}


BasicLayer::~BasicLayer () { 
   if (this->layerMatrix != nullptr) {
      delete this->getLayer();
   }
   if (this->getBias() != nullptr) {
      delete this->getBias();
   }
   if (this->getWeights() != nullptr) {
      delete this->getWeights();
   }
   if (this->getNext() != nullptr) {
		delete this->getNext();
	}
}

// broken function
BasicLayer::BasicLayer (const BasicLayer& b, bool copyNext) {
	if (b.getLayer() == nullptr) {
		this->layerMatrix = nullptr;
	} else {
		this->layerMatrix = new Matrix3D(*b.getLayer());
	}
	this->biasMatrixes = new Matrix3D*[1];
	this->biasMatrixes[0] = nullptr;
	if (b.getBias() != nullptr) {
		this->biasMatrixes[0] = new Matrix3D(*(b.getBias()));
	}
	this->weights = (WeightBase**) new BasicWeight*[1];
	this->weights[0] = nullptr;
	if (b.getWeights() != nullptr) {
		this->weights[0] = new BasicWeight(*(b.getWeights()));
	}

	this->next = (LayerBase**) new BasicLayer*[1];
	this->next[0] = nullptr;
	this->prev = (LayerBase**) new BasicLayer*[1];
	this->prev[0] = nullptr;

	if (copyNext) {
		const BasicLayer* bCurrent = &b;
		BasicLayer* thisCurrent = this;
		while (bCurrent->getNext() != nullptr) {
			bCurrent = bCurrent->getNext();
			thisCurrent->next = (LayerBase**) new BasicLayer*[1];
			thisCurrent->next[0] = new BasicLayer();
			if (bCurrent->getLayer() != nullptr) {
				thisCurrent->getNext()->setLayer(new Matrix3D (*bCurrent->getLayer()));
			}
			if (bCurrent->getBias() != nullptr) {
				thisCurrent->getNext()->setBias(new Matrix3D (*bCurrent->getBias()));
			}
			if (bCurrent->getWeights() != nullptr) {
				thisCurrent->getNext()->setWeights(new BasicWeight (*bCurrent->getWeights()));
			}
			thisCurrent->getNext()->setPrev(thisCurrent);
			thisCurrent = thisCurrent->getNext();
		}
	}
} 

BasicLayer* BasicLayer::getNext () const {
   return ((BasicLayer*) this->LayerBase::getNext(0));
}

BasicLayer* BasicLayer::getPrev () const {
   return ((BasicLayer*) this->LayerBase::getPrev(0));
}

BasicLayer* BasicLayer::getLast () {
   if (this->LayerBase::getNext(0) == nullptr) {
      return this;
   }
   return ((BasicLayer*) this->LayerBase::getNext(0))->getLast();
}

Matrix3D* BasicLayer::getLayer () const {
   return this->LayerBase::getLayer();
}

Matrix3D* BasicLayer::getBias () const {
   return this->LayerBase::getBias(0);
}

BasicWeight* BasicLayer::getWeights () const {
	return (BasicWeight*) this->LayerBase::getWeights(0);
}

void BasicLayer::setNext (BasicLayer* next) {
   if (this->getNext() != nullptr) {
      delete this->getNext();
   }
   this->next[0] = next;
} 

void BasicLayer::setPrev (BasicLayer* prev) {
   if (this->getPrev() != nullptr) {
      delete this->getPrev();
   }
   this->prev[0] = prev;
} 

void BasicLayer::setLayer (Matrix3D* layerMatrix) {
   if (this->getLayer() == nullptr) {
      this->layerMatrix = new Matrix3D (layerMatrix->getLength(), layerMatrix->getWidth(), layerMatrix->getHeight());
   }
   this->getLayer()->setMatrix(layerMatrix);
}

void BasicLayer::setBias (Matrix3D* biasMatrix) {
   if (this->getBias() == nullptr) {
      this->biasMatrixes[0] = new Matrix3D (biasMatrix->getLength(), biasMatrix->getWidth(), biasMatrix->getHeight());
   }
   this->getBias()->setMatrix(biasMatrix);
}

void BasicLayer::setWeights (BasicWeight* weights) {
   if (this->getWeights() != nullptr) {
      delete this->getWeights();
   }
   this->weights[0] = weights;
}

BasicLayer* BasicLayer::add (BasicLayer* layer) {
   if (this->getNext() == nullptr) {
      this->next[0] = layer;
   } else {
      this->next[0] = this->getNext()->add(layer);
   }
   return this;
}

BasicLayer* BasicLayer::add (Matrix3D* layerMatrix, Matrix3D* biasMatrix, BasicWeight* weights) {
   if (this->getNext() == nullptr) {
      this->next[0] = new BasicLayer (layerMatrix, nullptr, nullptr);
      this->getNext()->setPrev(this);
      if (this->getBias() == nullptr) {
         this->biasMatrixes[0] = new Matrix3D(this->getNext()->getLayer()->getLength(), this->getNext()->getLayer()->getWidth(), this->getNext()->getLayer()->getHeight());
         this->getBias()->randomize(-0.05, 0.05);
      } else {
         this->getBias()->setMatrix(biasMatrix);
      }
		
		this->weights[0] = this->newWeight(this, this->getNext());
      return this;
   }
   this->getNext()->add(layerMatrix, biasMatrix, weights);
   return this;
}

BasicWeight* BasicLayer::newWeight(BasicLayer* firstLayer, BasicLayer* secondLayer) {
	return new BasicWeight (
		firstLayer->getLayer()->getLength(), 
		firstLayer->getLayer()->getWidth(),
		firstLayer->getLayer()->getHeight(),
		secondLayer->getLayer()->getLength(),
		secondLayer->getLayer()->getWidth(),
		secondLayer->getLayer()->getHeight(),
		1);
}

void artificialIntelligence::classes::BasicLayer::calculateAndUpdateAllCPU () {
   if (this->getNext() == nullptr) {
      return;
   }
   this->calculateAndUpdateLayerCPU();
   this->getNext()->calculateAndUpdateAllCPU();
}

void BasicLayer::calculateAndUpdateLayerCPU () {
   Matrix3D* nextLayer = this->getNext()->getLayer();
   Matrix3D* outputs = new Matrix3D (nextLayer->getLength(), nextLayer->getWidth(), nextLayer->getHeight());
	outputs->setAll(0);
   if (isnan(*outputs->getData(0, 0, 0))) {
      std::cout << "null init";
      exit (0);
   }
   float activation = 0;
   for (int fl = 0; fl < this->getLayer()->getLength(); fl++) {
      for (int fw = 0; fw < this->getLayer()->getWidth(); fw++) {
         for (int fh = 0; fh < this->getLayer()->getHeight(); fh++) {
            for (int sl = 0; sl < nextLayer->getLength(); sl++) {
               for (int sw = 0; sw < nextLayer->getWidth(); sw++) {
                  for (int sh = 0; sh < nextLayer->getHeight(); sh++) {
                     activation = *this->getLayer()->getData(fl, fw, fh) * *this->getWeights()->getData(fl, fw, fh, sl, sw, sh) + *outputs->getData(sl, sw, sh);
                     outputs->insert(activation, sl, sw, sh);
                  }
               }
            }
         }
      }
   } 

   for (int sl = 0; sl < nextLayer->getLength(); sl++) {
      for (int sw = 0; sw < nextLayer->getWidth(); sw++) {
         for (int sh = 0; sh < nextLayer->getHeight(); sh++) {
            activation = sigmoid(*outputs->getData(sl, sw, sh) + *this->getBias()->getData(sl, sw, sh));
            outputs->insert(activation, sl, sw, sh);
         }
      }
   }

   this->getNext()->setLayer (outputs);
   delete outputs;
}

void BasicLayer::calculateAndUpdateAllGPUV2() {
	BasicLayer* currentLayer = this;
	Matrix3D* currentLayerMatrix = currentLayer->getLayer();

	long long numInputs = currentLayerMatrix->getSize() / sizeof(float);
	long long numOutputs = currentLayer->getNext()->getLayer()->getSize() / sizeof(float);
	long long numWeights = numInputs * numOutputs;
	long long numOutputsRemaining = numOutputs;
	long long outputIndex = 0;

	long long numBlocks = numOutputs > MAX_BLOCK_SIZE ? MAX_BLOCK_SIZE : numOutputs; 
	long long numThreads = 512;
	long long maxWeightIndex = currentLayer->getWeights()->getWeightMatrix()->getSize() / sizeof(float);
	long long numPerThread = std::ceil ((double)maxWeightIndex / (numBlocks * numThreads));
	long long sharedSize = numThreads * sizeof(float); 
	if (maxWeightIndex > numWeights) {
		maxWeightIndex = numWeights;
	}

	float* input = currentLayerMatrix->getArr();
	float* output = currentLayer->getNext()->getLayer()->getArr();
	float* current_input;
	float* current_output;
	gpuErrchk(hipMalloc((void **) &current_input, currentLayerMatrix->getSize()));
	gpuErrchk(hipMalloc((void **) &current_output, numOutputs * sizeof(float)));

	// streams for asynchronous
	hipStream_t stream1, stream2;
	hipStreamCreate ( &stream1); 
	hipStreamCreate ( &stream2); 
	
	BasicWeight* currentWeight = currentLayer->getWeights();
	long long currentWeightMatrixIndex = 0;
	long long weightsAddedLastSet = 0;
	long long weightsInCurrentKernelRun = 0;

	float* current_weights;
	float* next_weights;

	gpuErrchk(hipMalloc((void **) &current_weights, maxWeightIndex * sizeof(float)));
	gpuErrchk(hipMalloc((void **) &next_weights, maxWeightIndex * sizeof(float)));
	gpuErrchk(hipMemcpy(current_weights, currentWeight->getWeightMatrix(0)->getArr(), maxWeightIndex * sizeof(float), hipMemcpyHostToDevice));
	weightsInCurrentKernelRun = maxWeightIndex;
	weightsAddedLastSet = maxWeightIndex;
	currentWeightMatrixIndex++;
	
	int startingOutputID = 0;
	int nextOutputID = maxWeightIndex % currentWeight->getOutputSize();
	int numWeightsMatrixesLeft = std::ceil((float)numWeights / maxWeightIndex) - 1;

	gpuErrchk(hipMemcpy(current_input, input, currentLayerMatrix->getSize(), hipMemcpyHostToDevice)); 
	gpuErrchk(hipMemset(current_output, 0b00000000, numOutputs * sizeof(float)));

	int debugCounter = 0;

	numWeightsMatrixesLeft = std::ceil((float)numWeights / maxWeightIndex) - 1;
   while (currentLayer->getNext() != nullptr) {
		currentWeightMatrixIndex = 1;
		outputIndex = 0;
		startingOutputID = 0;
		numOutputsRemaining = numOutputs;
		nextOutputID = weightsAddedLastSet;
		
		bool weightsFinished = false;
		long long weightsUsed = 0;
		do {
			
			if (numWeightsMatrixesLeft >= 1){
				if (currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getSize() / sizeof(float) < maxWeightIndex) {
					maxWeightIndex = currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getSize() / sizeof(float);
				}
				gpuErrchk(hipMemcpyAsync(next_weights, currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getArr(), maxWeightIndex * sizeof(float), hipMemcpyHostToDevice));
				weightsAddedLastSet = maxWeightIndex;
				currentWeightMatrixIndex++;
				numWeightsMatrixesLeft -= 1;
			} 
			
			else { 
				if (currentLayer->getNext()->getNext() != nullptr) {
					int nextNumWeights = numOutputs * currentLayer->getNext()->getNext()->getLayer()->getSize() / sizeof(float);
					int nextMaxWeightIndex = currentLayer->getNext()->getWeights()->getWeightMatrix(0)->getSize() / sizeof(float);
					if (nextMaxWeightIndex > nextNumWeights) {
						nextMaxWeightIndex = nextNumWeights;
					}
					
					gpuErrchk(hipFree(next_weights));
					gpuErrchk(hipMalloc((void **) &next_weights, nextMaxWeightIndex * sizeof(float)));
					gpuErrchk(hipMemcpyAsync(next_weights, currentLayer->getNext()->getWeights()->getWeightMatrix(0)->getArr(), nextMaxWeightIndex * sizeof(float), hipMemcpyHostToDevice));
					currentWeightMatrixIndex = 1;
					numWeightsMatrixesLeft = std::ceil((float)nextNumWeights / nextMaxWeightIndex) - 1;
					weightsAddedLastSet = nextMaxWeightIndex;
				}
				weightsFinished = true;
			}
			
			long long helper = 0;

			do {
				if (numOutputsRemaining > 0) {
					// std::cout << "inside22\n";
					// std::cout << "numBlocks: " << numBlocks << '\n';
					// std::cout << "numOutputs: " << numOutputs << '\n';
					// std::cout << "numPerThread: " << numPerThread << '\n';
					// std::cout << "weightsInCurrentKernelRun: " << weightsInCurrentKernelRun << "\n";
					// std::cout << "numOutputsRemaining: " << numOutputsRemaining << '\n';
					// std::cout << "helper: " << helper << '\n';
					// std::cout << "weightsUsed: " << weightsUsed << "\n";
					// std::cout << "numWeightsMatrixesLeft: " << numWeightsMatrixesLeft << "\n";
					// std::cout << "weightsAddedLastSet: " << weightsAddedLastSet << "\n";
					// std::cout << "startingOutputID: " << startingOutputID << "\n\n";
					
					if (numOutputsRemaining - numBlocks < 0) {
						numBlocks = numOutputsRemaining;
					}

					artificialIntelligence::classes::calculateAndUpdateLayerGPU<<< numBlocks, numThreads, sharedSize, stream1 >>>(current_input, current_weights, current_output, numBlocks, numOutputs, numPerThread, weightsInCurrentKernelRun, helper, weightsUsed, startingOutputID);
					outputIndex += numBlocks;
					numOutputsRemaining -= numBlocks;
				}
				startingOutputID += numBlocks;
				helper += numBlocks;

			} while (numOutputsRemaining > 0);
			gpuErrchk(hipDeviceSynchronize());
			
			startingOutputID = nextOutputID % numOutputs;
			nextOutputID += weightsInCurrentKernelRun % numOutputs;
			numOutputsRemaining = numOutputs;

			weightsUsed += weightsInCurrentKernelRun;
			numBlocks = numOutputs > MAX_BLOCK_SIZE ? MAX_BLOCK_SIZE : numOutputs;
			weightsInCurrentKernelRun = weightsAddedLastSet;

			float* temp = current_weights;
			current_weights = next_weights;
			next_weights = temp;

		} while (!weightsFinished);
		
		gpuErrchk(hipMemcpy(output, current_output, numOutputs * sizeof(float), hipMemcpyDeviceToHost));
		// printArr(currentWeight->getWeightMatrix(currentWeightMatrixIndex - 1)->getArr(), 10);
		Matrix3D* bias = currentLayer->getBias();
		currentLayer = currentLayer->getNext();
		currentLayerMatrix = currentLayer->getLayer();
		currentWeight = currentLayer->getWeights();
		numInputs = currentLayerMatrix->getSize() / sizeof(float);

		if (currentLayer->getNext() != nullptr) {
			output = currentLayer->getNext()->getLayer()->getArr();
			numOutputs = currentLayer->getNext()->getLayer()->getSize() / sizeof(float);
			numWeights = numInputs * numOutputs;
			maxWeightIndex = currentLayer->getWeights()->getWeightMatrix()->getSize();
			numBlocks = numOutputs > MAX_BLOCK_SIZE ? MAX_BLOCK_SIZE : numOutputs;
			numThreads = 512; // arbitrary
			numPerThread = std::ceil ((double)maxWeightIndex / (numBlocks * numThreads));
			output = currentLayer->getNext()->getLayer()->getArr();
			gpuErrchk(hipFree(next_weights));
			gpuErrchk(hipMalloc((void **) &next_weights, maxWeightIndex * sizeof(float)));
			gpuErrchk(hipFree(current_output));
			gpuErrchk(hipMalloc((void **) &current_output, numOutputs * sizeof(float)));
			gpuErrchk(hipMemset(current_output, 0b00000000, numOutputs * sizeof(float))); 
		}
		
		*currentLayer->getLayer() += bias;
		sigmoid(currentLayer->getLayer(), false);
		gpuErrchk(hipFree(current_input));
		gpuErrchk(hipMalloc((void **) &current_input, currentLayerMatrix->getSize()));
		input = currentLayerMatrix->getArr();
		gpuErrchk(hipMemcpy(current_input, input, currentLayerMatrix->getSize(), hipMemcpyHostToDevice));

		debugCounter++;
	}
	gpuErrchk(hipFree(current_input));	
	gpuErrchk(hipFree(current_output));
	gpuErrchk(hipFree(current_weights));	
	gpuErrchk(hipFree(next_weights));
	gpuErrchk(hipStreamDestroy(stream1));
	gpuErrchk(hipStreamDestroy(stream2));
}

__global__ void artificialIntelligence::classes::calculateAndUpdateLayerGPU(float* nodeValues, float* weights, float* output, int inputSize, int outputSize, int numPerThread, long long maxWeightIndex, long long helperIndex, long long startingWeight, int startingOutputId) {
	extern __shared__ float sdata[];
	unsigned int tid = threadIdx.x;
	unsigned long long outputNodeId = (blockIdx.x + startingOutputId) % outputSize;
	unsigned int numThreads = blockDim.x;
	unsigned long long weightIndex = tid * outputSize + blockIdx.x + helperIndex;
	unsigned long long inputNodeId = 0;
	unsigned int gridSize = numThreads*outputSize;
	sdata[tid] = 0;

	while (weightIndex < maxWeightIndex) {
		inputNodeId = (weightIndex + startingWeight) / outputSize;
		sdata[tid] += nodeValues[inputNodeId] * weights[weightIndex];
		weightIndex += gridSize;
	}

	__syncthreads();

	for (unsigned int s=numThreads/2; s>0; s>>=1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}
	
	if (tid == 0) {
		output[outputNodeId] += sdata[0];
	}
}

Matrix3D* BasicLayer::calculateErrorCPU (Matrix3D* delta) {
	Matrix3D* currentLayerMatrix = this->getLayer();
	Matrix3D* error = new Matrix3D(currentLayerMatrix->getLength(), currentLayerMatrix->getWidth(), currentLayerMatrix->getHeight());
	for (int l = 0; l < currentLayerMatrix->getLength(); l++) {
		for (int w = 0; w < currentLayerMatrix->getWidth(); w++) {
			for (int h = 0; h < currentLayerMatrix->getHeight(); h++) {
				Matrix3D* outputMatrix = this->getNext()->getLayer();
				Matrix3D* weightedMatrix = new Matrix3D (delta->getLength(), delta->getWidth(), delta->getHeight());
				for (int l2 = 0; l2 < outputMatrix->getLength(); l2++) {
					for (int w2 = 0; w2 < outputMatrix->getWidth(); w2++) {
						for (int h2 = 0; h2 < outputMatrix->getHeight(); h2++) {
							weightedMatrix->insert(*this->getWeights()->getData(l, w, h, l2, w2, h2) * *delta->getData(l2, w2, h2), l2, w2, h2);
						}
					}
				}
				error->insert(weightedMatrix->sum(), l, w, h);
				delete weightedMatrix;
			}
		}
	}
	return error;
}

Matrix3D* BasicLayer::calculateErrorGPU (Matrix3D* delta) {
	BasicLayer* currentLayer = this;
	Matrix3D* currentLayerMatrix = currentLayer->getLayer();

	long long numInputs = currentLayerMatrix->getSize() / sizeof(float);
	long long numOutputs = currentLayer->getNext()->getLayer()->getSize() / sizeof(float);
	long long numWeights = numInputs * numOutputs;
	long long numInputsRemaining = numInputs;
	long long inputIndex = 0;
	long long numBlocks = numInputs > MAX_BLOCK_SIZE ? MAX_BLOCK_SIZE : numInputs; 
	long long numThreads = 512;
	long long maxWeightIndex = numBlocks * numOutputs;
	long long numPerThread = std::ceil ((double)maxWeightIndex / (numBlocks * numThreads));
	long long sharedSize = numThreads * sizeof(float); 
	if (maxWeightIndex > numWeights) {
		maxWeightIndex = numWeights;
	}
	Matrix3D* errorMatrix = new Matrix3D(currentLayer->getLayer()->getLength(), currentLayer->getLayer()->getWidth(), currentLayer->getLayer()->getHeight());
	float* error = errorMatrix->getArr();
	float* current_error;
	float* current_delta;
	gpuErrchk(hipMalloc((void **) &current_error, errorMatrix->getSize()));
	gpuErrchk(hipMalloc((void **) &current_delta, delta->getSize()));
	gpuErrchk(hipMemcpy(current_error, error, errorMatrix->getSize(), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(current_delta, delta->getArr(), delta->getSize(), hipMemcpyHostToDevice));

	hipStream_t stream1, stream2;
	hipStreamCreate ( &stream1); 
	hipStreamCreate ( &stream2); 
	
	BasicWeight* currentWeight = currentLayer->getWeights();
	long long matrixSize = currentWeight->getWeightMatrix()->getSize() / sizeof(float);
	long long currentWeightMatrixIndex = 0;
	long long weightsAddedLastSet = 0;
	long long weightsInCurrentKernelRun = 0;

	// std::cout << "Number of threads: " << numThreads << '\n';
	// std::cout << "Number of blocks: " << numBlocks << '\n';
	// std::cout << "Number per thread: " << numPerThread << '\n';
	// std::cout << "Number of bytes for shared storage: " << sharedSize << "\n";
	// std::cout << "Max array index: " << maxWeightIndex << "\n";
	// std::cout << "Max byte index: " << maxWeightIndex * sizeof(float) << "\n";
	// std::cout << "numLeftToAdd: " <<  numLeftToAdd << "\n";

	float* current_weights;
	float* next_weights;

	gpuErrchk(hipMalloc((void **) &current_weights, maxWeightIndex * sizeof(float)));
	gpuErrchk(hipMalloc((void **) &next_weights, maxWeightIndex * sizeof(float)));

	int weightsInCurrentMatrix = currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getSize() / sizeof(float);
	int weightsInBasicWeight = currentWeight->getSize();

	int numberOfWeightsToAdd = numBlocks * numOutputs;
	int toAdd = weightsInCurrentMatrix > numberOfWeightsToAdd ? numberOfWeightsToAdd : weightsInCurrentMatrix;

	int amountAdded = 0;
	int weightsAdded = 0;

	while (numberOfWeightsToAdd > 0) {
		toAdd = weightsInCurrentMatrix > numberOfWeightsToAdd ? numberOfWeightsToAdd : weightsInCurrentMatrix;
		// std::cout << "\ntoAdd: " <<  toAdd << "\n";
		// std::cout << "numberOfWeightsToAdd: " <<  numberOfWeightsToAdd << "\n";
		// std::cout << "weightsInCurrentMatrix: " <<  weightsInCurrentMatrix << "\n";
		// std::cout << "currentWeightMatrixIndex: " <<  currentWeightMatrixIndex << "\n";
		gpuErrchk(hipMemcpy(&current_weights[weightsAdded], &currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getArr()[amountAdded], toAdd * sizeof(float), hipMemcpyHostToDevice));
		if (toAdd == weightsInCurrentMatrix) {
			currentWeightMatrixIndex++;
			// std::cout << "inside1\n";
			numberOfWeightsToAdd -= toAdd;
			amountAdded = 0;
			weightsAdded += toAdd;
			if (weightsAdded < numWeights) {
				weightsInCurrentMatrix = currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getSize() / sizeof(float);
			}
		} else {
			amountAdded = toAdd;
			numberOfWeightsToAdd = 0;
			weightsInCurrentMatrix -= toAdd;
			weightsAdded += toAdd;
		}
	}
	weightsInCurrentKernelRun = weightsAdded;
	weightsAddedLastSet = weightsAdded;
	
	int startingInputID = 0;
	int numWeightsMatrixesLeft = std::ceil((float)numWeights / maxWeightIndex) - 1;

	numWeightsMatrixesLeft = std::ceil((float)numWeights / maxWeightIndex) - 1;
	inputIndex = 0;
	startingInputID = 0;
	numInputsRemaining = numInputs;
	bool weightsFinished = false;
	long long weightsUsed = 0;
	do {
		if (numInputsRemaining > 0) {
			numBlocks = (weightsUsed + weightsInCurrentKernelRun) * numInputs / numWeights - weightsUsed * numInputs / numWeights;
			// std::cout << "inside22\n";
			// std::cout << "numBlocks: " << numBlocks << '\n';
			// std::cout << "numInputs: " << numInputs << '\n';
			// std::cout << "numPerThread: " << numPerThread << '\n';
			// std::cout << "weightsInCurrentKernelRun: " << weightsInCurrentKernelRun << "\n";
			// std::cout << "numInputsRemaining: " << numInputsRemaining << '\n';
			// std::cout << "weightsUsed: " << weightsUsed << "\n";
			// std::cout << "startingInputID: " << startingInputID << "\n\n";

			if (numInputsRemaining - numBlocks < 0) {
				numBlocks = numInputsRemaining;
			}
			
			artificialIntelligence::classes::calculateError<<< numBlocks, numThreads, sharedSize, stream1 >>>(current_weights, current_delta, current_error, numInputs, numOutputs, numPerThread, weightsInCurrentKernelRun, numWeights, weightsUsed, startingInputID);
			inputIndex += numBlocks;
			numInputsRemaining -= numBlocks;
		}
		gpuErrchk(hipDeviceSynchronize());
		startingInputID += numBlocks;

		weightsUsed += weightsInCurrentKernelRun;
		weightsInCurrentKernelRun = weightsAddedLastSet;
		if (numWeights - weightsAdded > 0) {
			// std::cout << "here\n";
			// exit(0);
			// asynchronously add the next set of weights
			numBlocks = numInputsRemaining > MAX_BLOCK_SIZE ? MAX_BLOCK_SIZE : numInputsRemaining;
			numberOfWeightsToAdd = numBlocks * numOutputs;
			toAdd = weightsInCurrentMatrix > numberOfWeightsToAdd ? numberOfWeightsToAdd : weightsInCurrentMatrix;
			amountAdded = weightsAdded % (WEIGHT_MAX_SIZE);
			int weightCounter = 0;
			if (weightsInCurrentMatrix > 0) {
				weightsInCurrentKernelRun = numberOfWeightsToAdd;
				while (numberOfWeightsToAdd > 0) {
					toAdd = weightsInCurrentMatrix > numberOfWeightsToAdd ? numberOfWeightsToAdd : weightsInCurrentMatrix;
					// std::cout << "currentWeightMatrixIndex: " << currentWeightMatrixIndex << "\n";
					// std::cout << "amountAdded: " << amountAdded << "\n";
					// std::cout << "toAdd: " << toAdd << "\n";
					// std::cout << "weightsInCurrentMatrix: " << weightsInCurrentMatrix << "\n";
					// std::cout << "currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getArr()[amountAdded]: " << currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getArr()[amountAdded] << '\n';
					gpuErrchk(hipMemcpyAsync(&next_weights[weightCounter], &currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getArr()[amountAdded], toAdd * sizeof(float), hipMemcpyHostToDevice));
					if (toAdd == weightsInCurrentMatrix) {
						currentWeightMatrixIndex++;
						numberOfWeightsToAdd -= toAdd;
						amountAdded = 0;
						weightsAdded += toAdd;
						weightCounter += toAdd;
						if (weightsAdded < numWeights) {
							weightsInCurrentMatrix = currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getSize() / sizeof(float);
						}
					} else {
						numberOfWeightsToAdd = 0;
						weightsInCurrentMatrix -= toAdd;
						weightsAdded += toAdd;
					}
				}
			}
		}
		else {
			weightsFinished = true;
		}

		gpuErrchk(hipDeviceSynchronize());

		float* temp = current_weights;
		current_weights = next_weights;
		next_weights = temp;

	} while (!weightsFinished);

	gpuErrchk(hipMemcpy(error, current_error, numInputs * sizeof(float), hipMemcpyDeviceToHost));

	// :::: FREE ALL ALLOCATED MEMORY :::: //
	gpuErrchk(hipFree(current_error));	
	gpuErrchk(hipFree(current_delta));
	gpuErrchk(hipFree(current_weights));	
	gpuErrchk(hipFree(next_weights));
	gpuErrchk(hipStreamDestroy(stream1));
	gpuErrchk(hipStreamDestroy(stream2));
	return errorMatrix;
}

__global__ void artificialIntelligence::classes::calculateError(float* weights, float* delta, float* error, int inputSize, int outputSize, int numPerThread, long long maxWeightIndex, long long helperIndex, long long startingWeight, int startingInputID) {
	extern __shared__ float sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int numThreads = blockDim.x;
	unsigned long long inputNodeId = blockIdx.x + startingInputID;
	unsigned long long weightIndex = tid + blockIdx.x * outputSize;
	unsigned int gridSize = numThreads;
	int weightsToAddStart = outputSize * (blockIdx.x);
	int weightsToAddEnd = outputSize * (blockIdx.x + 1);

	sdata[tid] = 0;
	while (weightIndex >= weightsToAddStart && weightIndex < weightsToAddEnd) {
		sdata[tid] += weights[weightIndex] * delta[(startingWeight + weightIndex) % outputSize];
		weightIndex += gridSize;
	}

	__syncthreads();

	for (unsigned int s=numThreads/2; s>0; s>>=1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}
	
	if (tid == 0) {
		error[inputNodeId] += sdata[0];
	}
}

void BasicLayer::updateWeightsCPU (Matrix3D* delta, double learningRate) {
	Matrix3D* currentLayerMatrix = this->getLayer();
	for (int l = 0; l < currentLayerMatrix->getLength(); l++) {
		for (int w = 0; w < currentLayerMatrix->getWidth(); w++) {
			for (int h = 0; h < currentLayerMatrix->getHeight(); h++) {
				float inputValue = *currentLayerMatrix->getData(l, w, h);
				float value = 0;
				
				Matrix3D* weightMatrix = this->getNext()->getLayer();
				for (int l2 = 0; l2 < weightMatrix->getLength(); l2++) {
					for (int w2 = 0; w2 < weightMatrix->getWidth(); w2++) {
						for (int h2 = 0; h2 < weightMatrix->getHeight(); h2++) {
							value = *this->getWeights()->getData(l, w, h, l2, w2, h2) + inputValue * *delta->getData(l2, w2, h2) * learningRate;
							this->getWeights()->insertData(value, l, w, h, l2, w2, h2);
						}
					}
				}
			}
		}
	}
}

void BasicLayer::updateWeightsGPU (Matrix3D* delta, double learningRate) {
	BasicLayer* currentLayer = this;
	Matrix3D* currentLayerMatrix = currentLayer->getLayer();

	long long numInputs = currentLayerMatrix->getSize() / sizeof(float);
	long long numOutputs = currentLayer->getNext()->getLayer()->getSize() / sizeof(float);
	long long numWeights = numInputs * numOutputs;
	long long inputIndex = 0;
	long long numBlocks = numOutputs > MAX_BLOCK_SIZE ? MAX_BLOCK_SIZE : numOutputs; 
	long long numThreads = 512;
	long long maxWeightIndex = numBlocks * numOutputs;
	long long numPerThread = std::ceil ((double)maxWeightIndex / (numBlocks * numThreads));
	long long sharedSize = numThreads * sizeof(float); 
	if (maxWeightIndex > numWeights) {
		maxWeightIndex = numWeights;
	}
	
	hipStream_t stream1, stream2;
	hipStreamCreate ( &stream1); 
	hipStreamCreate ( &stream2);

	BasicWeight* currentWeight = currentLayer->getWeights();
	long long matrixSize = currentWeight->getWeightMatrix()->getSize() / sizeof(float);
	long long currentWeightMatrixIndex = 0;
	long long weightsInCurrentKernelRun = 0;
	
	int weightsInCurrentMatrix = currentWeight->getWeightMatrix(0)->getSize() / sizeof(float);

	Matrix3D* inputMatrix = currentLayer->getLayer();
	float* current_input;
	float* current_delta;
	gpuErrchk(hipMalloc((void **) &current_input, inputMatrix->getSize()));
	gpuErrchk(hipMalloc((void **) &current_delta, delta->getSize()));
	gpuErrchk(hipMemcpy(current_input, inputMatrix->getArr(), inputMatrix->getSize(), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(current_delta, delta->getArr(), delta->getSize(), hipMemcpyHostToDevice));
	

	float* current_weights;
	float* next_weights;
	gpuErrchk(hipMalloc((void **) &current_weights, currentWeight->getWeightMatrix(0)->getSize()));
	gpuErrchk(hipMalloc((void **) &next_weights, currentWeight->getWeightMatrix(0)->getSize()));
	gpuErrchk(hipMemcpy(current_weights, currentWeight->getWeightMatrix(0)->getArr(), currentWeight->getWeightMatrix(0)->getSize(), hipMemcpyHostToDevice));
	weightsInCurrentKernelRun = currentWeight->getWeightMatrix(0)->getSize() / sizeof(float);

	long long weightsUsed = 0;
	int startingInputId = 0;
	while ((numWeights - weightsUsed) != 0) {
		// std::cout << "inside22\n";
		// std::cout << "numBlocks: " << numBlocks << '\n';
		// std::cout << "numOutputs: " << numOutputs << '\n';
		// std::cout << "numPerThread: " << numPerThread << '\n';
		// std::cout << "weightsInCurrentKernelRun: " << weightsInCurrentKernelRun << "\n";
		// std::cout << "numOutputsRemaining: " << numOutputsRemaining << '\n';
		// std::cout << "weightsUsed: " << weightsUsed << "\n";
		// std::cout << "startingInputId: " << startingInputId << "\n\n";

		artificialIntelligence::classes::updateWeights<<<numBlocks, numThreads, sharedSize, stream1>>>(current_weights, current_delta, current_input, numInputs, numOutputs, numPerThread, weightsInCurrentKernelRun, numWeights, weightsUsed, startingInputId, learningRate);
		inputIndex += numBlocks;
		
		startingInputId = weightsUsed / numOutputs;

		weightsUsed += currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getSize() / sizeof(float);

		currentWeightMatrixIndex++;
		if ((numWeights - weightsUsed) != 0) {
			gpuErrchk(hipMemcpyAsync(next_weights, currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getArr(), currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getSize(), hipMemcpyHostToDevice));
			weightsInCurrentKernelRun = currentWeight->getWeightMatrix(currentWeightMatrixIndex)->getSize() / sizeof(float);
		}

		gpuErrchk(hipDeviceSynchronize());
		gpuErrchk(hipMemcpy(currentWeight->getWeightMatrix(currentWeightMatrixIndex - 1)->getArr(), current_weights, currentWeight->getWeightMatrix(currentWeightMatrixIndex - 1)->getSize(), hipMemcpyDeviceToHost));

		float* temp = current_weights;
		current_weights = next_weights;
		next_weights = temp;
	}

	gpuErrchk(hipFree(current_input));
	gpuErrchk(hipFree(current_delta));
	gpuErrchk(hipFree(current_weights));	
	gpuErrchk(hipFree(next_weights));
	gpuErrchk(hipStreamDestroy(stream1));
	gpuErrchk(hipStreamDestroy(stream2));
}

__global__ void artificialIntelligence::classes::updateWeights(float* weights, float* delta, float* input, int inputSize, int outputSize, int numPerThread, long long maxWeightIndex, long long helperIndex, long long startingWeight, int startingInputID, double learningRate) {
	extern __shared__ float sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int numThreads = blockDim.x;
	unsigned long long weightIndex = tid + numThreads * blockIdx.x;
	unsigned long long outputNodeId = (weightIndex + startingWeight) % outputSize;
	unsigned long long inputNodeId = (weightIndex + startingWeight) / outputSize;
	unsigned int gridSize = numThreads * gridDim.x;
	while (weightIndex < maxWeightIndex) {
		weights[weightIndex] += input[inputNodeId] * delta[outputNodeId] * learningRate;
		weightIndex += gridSize;
		inputNodeId = (weightIndex + startingWeight) / outputSize;
		outputNodeId = (weightIndex + startingWeight) % outputSize;
	}
}

int BasicLayer::print (bool printBias, bool printWeights, int depth) {
   if (this->layerMatrix != nullptr) {
      std::cout << "\n\nCurrent Index: " << depth << '\n';
      std::cout << "Layer Matrix: \n";
      this->getLayer()->printMatrix();
   } else {
      std::cout << "No layer found!\n";
      return depth;
   }
   if (printBias) {
      if (this->biasMatrixes[0] != nullptr) {
         std::cout << "Bias Matrix: \n";
         this->getBias()->printMatrix();
      } else {
         std::cout << "No biases found!\n";
      }
   }
   if (printWeights) {
      if (this->weights[0] != nullptr) {
			std::cout << "Weight Matrix: \n";
         this->getWeights()->print();
      } else {
         std::cout << "No weights found!\n";
      }
   }
   if (this->getNext() == nullptr) {
      return depth;
   }
   return this->getNext()->print(printBias, printWeights, depth + 1);
}  


void BasicLayer::toFile (std::ofstream* outputFile) {
	char* output = new char[sizeof(int) * 6];
   *outputFile << this->getLayer()->getLength() << ',' << this->getLayer()->getWidth() << ',' << this->getLayer()->getHeight() << '\n';

   if (this->biasMatrixes[0] == nullptr) {
      return;
   }
   *outputFile << this->getBias()->getLength() << ',' << this->getBias()->getWidth() << ',' << this->getBias()->getHeight() << '\n';
   for (int i = 0; i < this->getBias()->getLength(); i++) {
      for (int j = 0; j < this->getBias()->getWidth(); j++) {
         for (int k = 0; k < this->getBias()->getHeight(); k++) {
            *outputFile << *this->getBias()->getData(i, j, k) << ',';
         }
      }
   }

   outputFile->seekp((int) outputFile->tellp() - 1);
   outputFile->write("\n", 1);

   if (this->weights[0] == nullptr) {
      return;
   }

   *outputFile << this->getLayer()->getLength() << ',' << this->getLayer()->getWidth() << ',' << this->getLayer()->getHeight() << ',';
   *outputFile << this->getBias()->getLength() << ',' << this->getBias()->getWidth() << ',' << this->getBias()->getHeight() << '\n';

	int currentWeightMatrix = 0;
	float* weights;
	
	while (this->getWeights()->getWeightMatrix(currentWeightMatrix) != nullptr) {
		int size = this->getWeights()->getWeightMatrix(currentWeightMatrix)->getSize() + sizeof(float);
		char* output = new char[size];
		char* ptr = output;

		weights = this->getWeights()->getWeightMatrix(currentWeightMatrix)->getArr();
		for (int i = 0, cc = this->getWeights()->getWeightMatrix(currentWeightMatrix)->getSize() / sizeof(float); i < cc; i++) {
			memcpy(ptr, &weights[i], sizeof(float));
			ptr += sizeof(float);
		}
		outputFile->write(output, size);
		currentWeightMatrix++;

		free(output);

	}

   outputFile->seekp((int) outputFile->tellp() - 1);
   outputFile->write("\n", 1);

   if (this->getNext() == nullptr) {
      return;
   }
   this->getNext()->toFile(outputFile);
}


BasicLayer* BasicLayer::loadFromFile (std::ifstream* inputFile, BasicLayer* prev) {
	std::cout << "Loading layer from file\n";
   BasicLayer* layer = new BasicLayer ();
   std::string line;
   getline (*inputFile, line);
   std::stringstream lineStream;
   lineStream << line;
   std::string value;
   getline(lineStream, value, ',');
	std::cout << "v1: " << value << '\n';
   int layerLength = stoi(value);
   getline(lineStream, value, ',');
	std::cout << "v2: " << value << '\n';
   int layerWidth = stoi(value);
   getline(lineStream, value, ',');
	std::cout << "v3: " << value << '\n';
   int layerHeight = stoi(value);
   Matrix3D* layerMatrix = new Matrix3D (layerLength, layerWidth, layerHeight);
   layer->layerMatrix = layerMatrix;
   layer->prev[0] = prev;

   lineStream.str(std::string());
   lineStream.clear();
   getline (*inputFile, line);
   lineStream << line;

   if (inputFile->eof()) {
		layer->biasMatrixes = new Matrix3D*[1];
   	layer->weights = (WeightBase**) new BasicWeight*[1];
      return layer;
   }

   getline(lineStream, value, ',');
   int biasLength = stoi(value);
   getline(lineStream, value, ',');
   int biasWidth = stoi(value);
   getline(lineStream, value, ',');
   int biasHeight = stoi(value);
   Matrix3D* biasMatrix = new Matrix3D (biasLength, biasWidth, biasHeight);
   layer->biasMatrixes[0] = biasMatrix;

   lineStream.str(std::string());
   lineStream.clear();
   getline (*inputFile, line);
   lineStream << line;
   for (int i = 0; i < layer->getBias()->getLength(); i++) {
      for (int j = 0; j < layer->getBias()->getWidth(); j++) {
         for (int k = 0; k < layer->getBias()->getHeight(); k++) {
            std::getline(lineStream, value, ',');
            layer->getBias()->insert (stod(value), i, j, k);
         }
      }
   }

   getline (*inputFile, line);

   if (inputFile->eof()) {
		layer->weights = (WeightBase**) new BasicWeight*[1];
      return layer;
   }

   BasicWeight* weights = new BasicWeight (
      layer->getLayer()->getLength(), 
      layer->getLayer()->getWidth(), 
      layer->getLayer()->getHeight(), 
      layer->getBias()->getLength(), 
      layer->getBias()->getWidth(), 
      layer->getBias()->getHeight(),
		0
   );

   
   lineStream.str(std::string());
   lineStream.clear();
	
	std::cout << "Inserting weights\n";

	int currentWeightMatrix = 0;
	while (weights->getWeightMatrix(currentWeightMatrix) != nullptr) {
		inputFile->read((char*) weights->getWeightMatrix(currentWeightMatrix)->getArr(), weights->getWeightMatrix(currentWeightMatrix)->getSize());
		currentWeightMatrix++;
	}
	getline(*inputFile, line);

	std::cout << "Finished weights\n";

   layer->weights[0] = weights;
   layer->next[0] = BasicLayer::loadFromFile (inputFile, layer);

   return layer;
}